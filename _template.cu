#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <memory>

#include <hip/hip_runtime.h>

__global__ void kernel(int N, int* A, int* B, int* C){
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < N) {
		C[tId] = A[tId] + B[tId];
	}
}

void printVector(int* vec, int size) {
	for (int i=0; i<size; i++) {
		printf("%d", vec[i]);
	}
	printf("\n");
}

int main(int argc, char **argv){
	hipError_t err = hipSuccess;
	int N = 256;
	int Nblocks = 1;
	int Nthreads = 256;
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* A = new int[N];
	int* B = new int[N];
	int* C = new int[N];
	size_t size = N * sizeof(int);

	for(int i = 0; i < N ;i++){
		A[i] = 1;
		B[i] = 2;
	}

	printVector(A, N);
	printVector(B, N);

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	kernel<<<Nblocks, Nthreads>>>(N, d_A, d_B, d_C);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printVector(C, N);

	delete[] A;
	delete[] B;
	delete[] C;

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
