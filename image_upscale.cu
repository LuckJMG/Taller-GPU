#include "hip/hip_runtime.h"
#include <stdio.h>
#define STB_IMAGE_IMPLEMENTATION
#include "attachments/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "attachments/stb_image_write.h"

#include <hip/hip_runtime.h>

__global__ void kernel(unsigned char* d_img, int width, int height, int channels, unsigned char* out_img){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
		int idx_tl = (2*y * width + 2*x) * channels;
		int idx_tr = (2*y * width + 2*x + 1) * channels;
		int idx_bl = ((2*y + 1) * width + 2*x) * channels;
		int idx_br = ((2*y + 1) * width + 2*x + 1) * channels;

		for (int c = 0; c < channels; c++) {
			out_img[idx_tl + c] = d_img[idx + c];
			out_img[idx_tr + c] = d_img[idx + c];
			out_img[idx_bl + c] = d_img[idx + c];
			out_img[idx_br + c] = d_img[idx + c];
		}
    }
}

int main(int argc, char **argv){
	// Cargar imagen
	int width, height, channels;
	unsigned char* img = stbi_load("attachments/image.jpg", &width, &height, &channels, 0);
	if (!img) {
		printf("Error at loading image.jpg\n");
		return 1;
	}

	// Configuración GPU
	hipError_t err = hipSuccess;
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
	printf("Grid: %d x %d, Block: %d x %d\n", grid.x, grid.y, block.x, block.y);

	// Inicializar valores
	size_t size = width * height * channels * sizeof(unsigned char);
	unsigned char* d_img = NULL;
	unsigned char* d_out_img = NULL;

	// Asignar memoria en GPU
	hipMalloc((void**)&d_img, size);
	hipMalloc((void**)&d_out_img, 4*size);
	hipMemcpy(d_img, img, size, hipMemcpyHostToDevice);  // Copiar imagen a GPU

	// Procesar imagen
	kernel<<<grid, block>>>(d_img, width, height, channels, d_out_img);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	unsigned char* out_img = (unsigned char*)malloc(4*size);
	hipMemcpy(out_img, d_out_img, 4*size, hipMemcpyDeviceToHost);  // Copiar imagen procesada de la GPU

	// Guardar imagen procesada
	if (!stbi_write_png("processed.jpg", 2*width, 2*height, channels, out_img, 2*width * channels)) {
		printf("Error at saving processed.jpg\n");
	} else {
		printf("Inverted image saved at processed.jpg\n");
	}

	// Limpiar memoria
	hipFree(d_img);
	stbi_image_free(img);
	delete d_out_img;

	return 0;
}
