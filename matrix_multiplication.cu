#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int N, int* A, int* B, int* C){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < N) {
		int idx = y*N + x;
		C[idx] = 0;
		for (int i = 0; i < N; i++) {
			int idA = (i*N) + x;
			int idB = (y*N) + i;
			C[idx] += A[idA]*B[idB];
		}
    }
}

void printMatrix(int* matrix, int size) {
	for (int row=0; row < size; row++) {
		for (int col=0; col < size; col++) {
			int id = row*size + col;
			printf("%d ", matrix[id]);
		}
		printf("\n");
	}
}

int main(int argc, char **argv){
	// Configuración GPU
	hipError_t err = hipSuccess;
	int N = 4;
	dim3 block(16, 16);
	dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	printf("Grid: %d x %d, Block: %d x %d\n", grid.x, grid.y, block.x, block.y);

	// Inicializar Vectores
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* A = new int[N*N];
	int* B = new int[N*N];
	int* C = new int[N*N];
	size_t size = N * N * sizeof(int);

	for(int i = 0; i < N*N ;i++){
		A[i] = 1;
		B[i] = 2;
	}

	printMatrix(A, N);
	printMatrix(B, N);

	// Asignar memoria en la GPU
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// Copiar inputs a GPU
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	// Correr kernel en la GPU
	kernel<<<grid, block>>>(N, d_A, d_B, d_C);
	hipDeviceSynchronize();  // Esperar a que todos los threads terminen
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copiar output de la GPU
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printMatrix(C, N);

	// Limpiar memoria
	delete[] A;
	delete[] B;
	delete[] C;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
