#include <hip/hip_runtime.h>

__global__ void kernel(int N, int* A, int* B, int* C){
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < N) {
		C[tId] = A[tId] + B[tId];
	}
}

void printVector(int* vec, int size) {
	for (int i=0; i<size; i++) {
		printf("%d ", vec[i]);
	}
	printf("\n");
}

int main(int argc, char **argv){
	// Configuración GPU
	hipError_t err = hipSuccess;
	int N = 256;
	int Nblocks = 1;
	int Nthreads = 256;

	// Inicializar Vectores
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* A = new int[N];
	int* B = new int[N];
	int* C = new int[N];
	size_t size = N * sizeof(int);

	for(int i = 0; i < N ;i++){
		A[i] = 1;
		B[i] = 2;
	}

	printVector(A, N);
	printVector(B, N);

	// Asignar memoria en la GPU
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// Copiar inputs a GPU
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	// Correr kernel en la GPU
	kernel<<<Nblocks, Nthreads>>>(N, d_A, d_B, d_C);
	hipDeviceSynchronize();  // Esperar a que todos los threads terminen
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copiar output de la GPU
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printVector(C, N);

	// Limpiar memoria
	delete[] A;
	delete[] B;
	delete[] C;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
